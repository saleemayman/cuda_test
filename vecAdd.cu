#include "hip/hip_runtime.h"

#include </usr/local/cuda/include/hip/hip_runtime.h>
#include "lbm_header.h"

extern "C" __global__ void vecAdd(int temp, float *a, float *b, float *c, size_t size)
{
	int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx_z = threadIdx.z + blockDim.z * blockIdx.z;

	int idx_xy = idx_y * (blockDim.x * gridDim.x) + idx_x;
	int idx = idx_z * (blockDim.x * gridDim.x + blockDim.y * gridDim.y) + idx_xy;

	// try shared memory allocation as per lbm_beta.cu
	__shared__ T dd_buf[1][LOCAL_WORK_GROUP_SIZE];
	extern __shared__ T *dd_buf_lid;


	if(idx < size)
	{
		dd_buf[1][idx] = a[idx];
		dd_buf_lid = &dd_buf[1][idx];

		//c[idx] = a[idx] + b[idx];
		c[idx] = *dd_buf_lid;
		//c[idx] = (T)(DOMAIN_CELLS + DOMAIN_CELLS_X + DOMAIN_CELLS_Y);
	}
}